/*
 * engineSystem_cuda.cu
 *
 *  Created on: May 28, 2016
 *      Author: asilkaratas
 */

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

extern "C"
{
    void cudaGLInit(int argc, char **argv)
    {
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        findCudaGLDevice(argc, (const char **)argv);
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsNone));
    }

    void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
    }

    void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
    {
        void *ptr;
        checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
                                                             *cuda_vbo_resource));
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
    }

    void copyArrayFromDevice(void *host, const void *device,
                             struct hipGraphicsResource **cuda_vbo_resource, int size)
    {
        if (cuda_vbo_resource)
        {
            device = mapGLBufferObject(cuda_vbo_resource);
        }

        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

        if (cuda_vbo_resource)
        {
            unmapGLBufferObject(*cuda_vbo_resource);
        }
    }
}

