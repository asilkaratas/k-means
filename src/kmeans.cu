#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kmeans.h"
#include <cfloat>

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;

    return ++n;
}

__host__ __device__ inline static
float calculateDistance(int    numObjs,
                    int    numClusters,
                    float *objects,
                    float *clusters,
                    int    objectId,
                    int    clusterId)
{
	float distance = (objects[objectId] - clusters[clusterId]) *
					 (objects[objectId] - clusters[clusterId]) +
					 (objects[numObjs * 1 + objectId] - clusters[numClusters * 1 + clusterId]) *
					 (objects[numObjs * 1 + objectId] - clusters[numClusters * 1 + clusterId]) +
					 (objects[numObjs * 2 + objectId] - clusters[numClusters * 2 + clusterId]) *
					 (objects[numObjs * 2 + objectId] - clusters[numClusters * 2 + clusterId]);

	return distance;
}

__global__ static
void findNearestCluster(int numObjs,
                          int numClusters,
                          float *objects,
                          float *deviceClusters,
                          int *membership,
                          int *intermediates)
{
    extern __shared__ unsigned int membershipChanged[];

    membershipChanged[threadIdx.x] = 0;

    int objectId = blockDim.x * blockIdx.x + threadIdx.x;

    if (objectId < numObjs) {
        int   index, i;
        float dist, minDist = FLT_MAX;

        for (i=0; i<numClusters; i++) {
            dist = calculateDistance(numObjs, numClusters, objects, deviceClusters, objectId, i);
            if (dist < minDist) {
            	minDist = dist;
                index = i;
            }
        }

        if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

        membership[objectId] = index;

        __syncthreads();

        //calculating how many membership changed in this block.
        //warning! blockDim.x must be power of 2
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
        	//printf("intermediates1:blockIdx.x:%d membershipChanged[0]:%d\n",blockIdx.x, membershipChanged[0]);
            intermediates[blockIdx.x] = membershipChanged[0];
        }
    }
}

__global__ static
void computeDelta(int *deviceIntermediates,
                   int numIntermediates,
                   int numIntermediates2,
                   int startIndex,
                   int *total)
{

    extern __shared__ unsigned int intermediates[];

/*
    if (threadIdx.x == 0) {
    	printf("\nstartIndex:%d:%d\n", startIndex, numIntermediates);
    }
*/
    int objectId = blockDim.x * blockIdx.x + threadIdx.x;
    //copy into shared memory
    intermediates[threadIdx.x] = (objectId < numIntermediates) ? deviceIntermediates[objectId] : 0;

    __syncthreads();

    //calculate how many membership changed. (Sum of block sums)
    //warning! numIntermediates2 must be power of 2.
    for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
    	//total[0] = intermediates[0];
    	//printf("intermediates:%d\n", intermediates[0]);
    	total[0] += intermediates[0];
    }
}


float** kmeans(float **objects,
                   int     numObjs,
                   int     numClusters,
                   float   threshold,
                   int    *membership,
                   int    *loop_iterations)
{
    int      i, j, index, loop=0;
    int     *newClusterSize;
    float    delta;
    float  **dimObjects;
    float  **clusters;
    float  **dimClusters;
    float  **newClusters;

    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;
    int *deviceIntermediates;
    int *deviceIntermediatesLast;
    int *deviceTotal;


    malloc2D(dimObjects, 3, numObjs, float);
    for (i = 0; i < 3; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j][i];
        }
    }

    malloc2D(dimClusters, 3, numClusters, float);
    for (i = 0; i < 3; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }

    for (i=0; i<numObjs; i++) membership[i] = -1;


    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    malloc2D(newClusters, 3, numClusters, float);
    memset(newClusters[0], 0, 3 * numClusters * sizeof(float));


    const unsigned int numThreadsPerClusterBlock = 1024;
    const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
    const unsigned int clusterBlockSharedDataSize = numThreadsPerClusterBlock * sizeof(unsigned int);
    //const unsigned int clusterBlockSharedDataSize = 49152;//49152
    //printf("sharedSize:%d\n", sizeof(unsigned char));

    const unsigned int numReductionThreadsMin = 1024;
    const unsigned int numReductionThreadsMax = nextPowerOfTwo(numClusterBlocks);
    const unsigned int numReductionThreads = numReductionThreadsMax < numReductionThreadsMin ? numReductionThreadsMax : numReductionThreadsMin;
    const unsigned int reductionBlockSharedDataSize = numReductionThreads * sizeof(unsigned int);
    const unsigned int reductionRound = (numClusterBlocks + numReductionThreads - 1)/numReductionThreads;//division takes floor.

    printf("numThreadsPerClusterBlock:%d \n", numThreadsPerClusterBlock);
    printf("clusterBlockSharedDataSize:%d \n", clusterBlockSharedDataSize);
    printf("numClusterBlocks:%d \n\n", numClusterBlocks);

    printf("numReductionThreadsMin:%d \n", numReductionThreadsMin);
    printf("numReductionThreadsMax:%d \n", numReductionThreadsMax);
    printf("numReductionThreads:%d \n", numReductionThreads);
    printf("reductionBlockSharedDataSize:%d \n", reductionBlockSharedDataSize);
    printf("reductionRound:%d \n\n", reductionRound);

    checkCuda(hipMalloc(&deviceObjects, numObjs*3*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*3*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&deviceIntermediates, numReductionThreadsMax*sizeof(unsigned int)));
    //checkCuda(hipMemset(&deviceIntermediates, 0, numReductionThreadsMax*sizeof(unsigned int)));
    checkCuda(hipMalloc(&deviceTotal, sizeof(unsigned int)));


    checkCuda(hipMemcpy(deviceObjects, dimObjects[0], numObjs*3*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership, numObjs*sizeof(int), hipMemcpyHostToDevice));

    do {
        checkCuda(hipMemcpy(deviceClusters, dimClusters[0], numClusters*3*sizeof(float), hipMemcpyHostToDevice));

        findNearestCluster <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numObjs, numClusters, deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

        hipDeviceSynchronize(); checkLastCudaError();

        delta = 0;
        checkCuda(hipMemcpy(deviceTotal, &delta, sizeof(int), hipMemcpyHostToDevice));
        for(i = 0; i < reductionRound; ++i)
        {
        	int numIntermediates = (i < reductionRound - 1) || (numClusterBlocks%numReductionThreads==0) ? numReductionThreads : (numClusterBlocks - i*numReductionThreads)%numReductionThreads;
        	//printf("numIntermediates:%d\n", numIntermediates);
        	computeDelta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
        	            (deviceIntermediates, numIntermediates, numReductionThreads, i * numReductionThreads, deviceTotal);

        	hipDeviceSynchronize(); checkLastCudaError();
        }

        int d;
		checkCuda(hipMemcpy(&d, deviceTotal, sizeof(int), hipMemcpyDeviceToHost));
		delta = (float)d;

		//printf("d:%d\n\n", d);


/*
        computeDelta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
            (deviceIntermediates, numClusterBlocks, numReductionThreads);

        hipDeviceSynchronize(); checkLastCudaError();

        int d;
        checkCuda(hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost));
        delta = (float)d;
*/
        checkCuda(hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost));

        for (i=0; i<numObjs; i++) {
            index = membership[i];

            newClusterSize[index]++;
            for (j=0; j<3; j++)
            {
            	newClusters[j][index] += objects[i][j];
            }

        }

        for (i=0; i<numClusters; i++) {
            for (j=0; j<3; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;
            }
            newClusterSize[i] = 0;
        }

        delta /= numObjs;
    } while (delta > threshold && ++loop < 100);

    *loop_iterations = loop;

    printf("delta:%.16f\n", delta);
    malloc2D(clusters, numClusters, 3, float);

    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < 3; j++) {
            clusters[i][j] = dimClusters[j][i];
        }
    }

    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));
    checkCuda(hipFree(deviceIntermediates));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}

